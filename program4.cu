#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <random>
#include <cstdint>
#include <iostream>
#include <cstring>

#define MATRIX_SIZE 1024

void fill_matrix(float* matrix, uint64_t n);
void print_matrix(float* matrix, uint64_t n);

// Uses 1 block with a 1D dimension; assumes total number of threads = N
// Has 2xNxNxN total floating-point operations
__global__ void gpu_basic_mm(float* matrix1, float* matrix2, float* result, uint64_t n)
{
    // Divide threads into indices (assuming 1D blocks)
    //int num_threads = gridDim.x * blockDim.x;
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

    // Assume num threads = n
    for (int row = 0; row < n; ++row)
    {
        float sum = 0;
        for (int item = 0; item < n; ++item)
        {
            sum += matrix1[row * n + item] * matrix2[item * n + thread_index];
        }
        result[thread_index * n + row] = sum;
    }
}

// Assumes blockDim = NxA and threadDim = B where A * B = N
// Has 2xNxNxN total floating-point operations
__global__ void gpu_better_mm(float* matrix1, float* matrix2, float* result, uint64_t n)
{
    int row = blockIdx.x;
    int column = blockIdx.y * blockDim.y + threadIdx.x;

    // Assume num threads = n * n
    float sum = 0;
    for (int item = 0; item < n; ++item)
    {
        sum += matrix1[row * n + item] * matrix2[item * n + column];
    }
    result[column * n + row] = sum;
}

__global__ void gpu_better_transpose_mm(float* matrix1, float* matrix2, float* result, uint64_t n)
{
    // int r = blockIdx.x;
    // if (threadIdx.x == 0 && blockIdx.y == 0)
    // {
    //     // Transpose the second matrix
    //     int start = 1;

    //     for (int c = start++; c < n; ++c)
    //     {
    //         int temp = c * n + r;
    //         matrix2[c * n + r] = matrix2[r * n + c];
    //         matrix2[r * n + c] = temp;
    //     }
    // }

    // int row = blockIdx.x;
    // int column = blockIdx.y * blockDim.y + threadIdx.x;

    // __shared__ float shared_matrix2[MATRIX_SIZE * MATRIX_SIZE];
    // __shared__ float shared_matrix1[MATRIX_SIZE * MATRIX_SIZE];

    // shared_matrix1[column * n + row] = matrix1[column * n + row];
    // shared_matrix2[column * n + row] = matrix2[column * n + row];

    // // Assume num threads = n * n
    // float sum = 0;
    // for (int item = 0; item < n; ++item)
    // {
    //     sum += shared_matrix1[row * n + item] * shared_matrix2[item * n + column];
    // }
    // result[column * n + row] = sum;
}

int main()
{
    // Size of matrices
    uint64_t n = MATRIX_SIZE;

    float* m1 = new float[n * n];
    float* m2 = new float[n * n];
    float* result = new float[n * n];

    float* g_m1;
    float* g_m2;
    float* g_result;
    hipMalloc(reinterpret_cast<void**>(&g_m1), n * n * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&g_m2), n * n * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&g_result), n * n * sizeof(float));

    fill_matrix(m1, n);
    fill_matrix(m2, n);

    hipError_t code = hipPeekAtLastError();
    if (code != hipSuccess)
    {
        printf("Allocation Error: %s\n", hipGetErrorString(code));
    }

    //printf("-----m1-----\n");
    //print_matrix(m1, n);

    //printf("-----m2-----\n");
    //print_matrix(m2, n);


    // Timer start including memcpy operations
    //auto start = std::chrono::system_clock::now();

    hipMemcpy(g_m1, m1, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g_m2, m2, n * n * sizeof(float), hipMemcpyHostToDevice);

    code = hipPeekAtLastError();
    if (code != hipSuccess)
    {
        printf("Memcpy Error: %s\n", hipGetErrorString(code));
    }

    // Timer start excluding memcpy operations
    clock_t start = clock();

    // has 2xnxnxn total floating-point operations

    // <<<A, B>>> where A * B = N
    gpu_basic_mm<<<32, 32>>>(g_m1, g_m2, g_result, n);
    //gpu_better_mm<<<dim3(256, 4), 1024, 2 * n * n>>>(g_m1, g_m2, g_result, n);
    hipDeviceSynchronize();

    // Timer end excluding memcpy operations
    float elapsed_seconds = (float)(clock() - start) / CLOCKS_PER_SEC;

    code = hipPeekAtLastError();
    if (code != hipSuccess)
    {
        printf("Kernel Error: %s\n", hipGetErrorString(code));
    }

    hipMemcpy(result, g_result, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Timer end including memcpy operations
    //float elapsed_seconds = (std::chrono::system_clock::now() - start).count();


    //printf("-----result-----\n");
    //print_matrix(result, n);

    code = hipPeekAtLastError();
    if (code != hipSuccess)
    {
        printf("Memcpy Error 2: %s\n", hipGetErrorString(code));
    }

    float flops = (2 * n * n * n) / elapsed_seconds;
    std::cout << "Operations: " << (2 * n * n * n) << std::endl;
    //printf("Operations: %lld\n", (2 * n * n * n));
    printf("Seconds: %f\n", elapsed_seconds);
    std::cout << "FLOPS for gpu_basic_mm() at size " << n << " matrices = " << flops << std::endl;
    //printf("FLOPS for gpu_basic_mm() at size %lld matrices = %f", n, flops);


    hipFree(m1);
    hipFree(m2);
    hipFree(g_result);

    delete[] m1;
    delete[] m2;
    delete[] result;

}

void fill_matrix(float* matrix, uint64_t n)
{
    std::uniform_real_distribution<float> distribution(2, 100);
    std::default_random_engine generator;

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            matrix[i * n + j] = distribution(generator);
        }
    }
}

void print_matrix(float* matrix, uint64_t n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f\t", matrix[i * n + j]);
        }
        printf("\n");
    }
}
